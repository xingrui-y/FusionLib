#include "hip/hip_runtime.h"
#include "pose_estimator.h"
#include "vector_math.h"
#include "cuda_utils.h"
#include "reduce_sum.h"
#include <thrust/device_vector.h>

namespace fusion
{

struct RgbReduction
{
    __device__ bool find_corresp(int &x, int &y)
    {
        float4 pt = last_vmap.ptr(y)[x];
        if (pt.w < 0 || isnan(pt.x))
            return false;

        i_l = last_image.ptr(y)[x];
        if (!isfinite(i_l))
            return false;

        p_transformed = pose(make_float3(pt));
        u0 = p_transformed.x / p_transformed.z * fx + cx;
        v0 = p_transformed.y / p_transformed.z * fy + cy;
        if (u0 >= 2 && u0 < cols - 2 && v0 >= 2 && v0 < rows - 2)
        {
            i_c = interp2(curr_image, u0, v0);
            dx = interp2(dIdx, u0, v0);
            dy = interp2(dIdy, u0, v0);

            return (dx > 2 || dy > 2) && isfinite(i_c) && isfinite(dx) && isfinite(dy);
        }

        return false;
    }

    __device__ float interp2(cv::cuda::PtrStep<float> image, float &x, float &y)
    {
        int u = floor(x), v = floor(y);
        float coeff_x = x - u, coeff_y = y - v;
        return (image.ptr(v)[u] * (1 - coeff_x) + image.ptr(v)[u + 1] * coeff_x) * (1 - coeff_y) +
               (image.ptr(v + 1)[u] * (1 - coeff_x) + image.ptr(v + 1)[u + 1] * coeff_x) * coeff_y;
    }

    __device__ void compute_jacobian(int &k, float *sum)
    {
        int y = k / cols;
        int x = k - y * cols;

        bool corresp_found = find_corresp(x, y);
        float row[7] = {0, 0, 0, 0, 0, 0, 0};

        if (corresp_found)
        {
            float3 left;
            float z_inv = 1.0 / p_transformed.z;
            left.x = dx * fx * z_inv;
            left.y = dy * fy * z_inv;
            left.z = -(left.x * p_transformed.x + left.y * p_transformed.y) * z_inv;
            row[6] = i_l - i_c;

            *(float3 *)&row[0] = left;
            *(float3 *)&row[3] = cross(p_transformed, left);
        }

        int count = 0;
#pragma unroll
        for (int i = 0; i < 7; ++i)
#pragma unroll
            for (int j = i; j < 7; ++j)
                sum[count++] = row[i] * row[j];

        sum[count] = (float)corresp_found;
    }

    __device__ __forceinline__ void operator()()
    {
        float sum[29] = {0, 0, 0, 0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0};

        float val[29];
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
        {
            compute_jacobian(i, val);
#pragma unroll
            for (int j = 0; j < 29; ++j)
                sum[j] += val[j];
        }

        BlockReduce<float, 29>(sum);

        if (threadIdx.x == 0)
#pragma unroll
            for (int i = 0; i < 29; ++i)
                out.ptr(blockIdx.x)[i] = sum[i];
    }

    int cols, rows, N;
    float u0, v0;
    DeviceMatrix3x4 pose;
    float fx, fy, cx, cy, invfx, invfy;
    cv::cuda::PtrStep<float4> point_cloud, last_vmap;
    cv::cuda::PtrStep<float> last_image, curr_image;
    cv::cuda::PtrStep<float> dIdx, dIdy;
    cv::cuda::PtrStep<float> out;
    float3 p_transformed, p_last;

private:
    float i_c, i_l, dx, dy;
};

__global__ void rgb_reduce_kernel(RgbReduction rr)
{
    rr();
}

void rgb_reduce(const cv::cuda::GpuMat &curr_intensity,
                const cv::cuda::GpuMat &last_intensity,
                const cv::cuda::GpuMat &last_vmap,
                const cv::cuda::GpuMat &curr_vmap,
                const cv::cuda::GpuMat &intensity_dx,
                const cv::cuda::GpuMat &intensity_dy,
                cv::cuda::GpuMat &sum,
                cv::cuda::GpuMat &out,
                const Sophus::SE3d &pose,
                const IntrinsicMatrix K,
                float *jtj, float *jtr,
                float *residual)
{
    int cols = curr_intensity.cols;
    int rows = curr_intensity.rows;

    RgbReduction rr;
    rr.cols = cols;
    rr.rows = rows;
    rr.N = cols * rows;
    rr.last_image = last_intensity;
    rr.curr_image = curr_intensity;
    rr.point_cloud = curr_vmap;
    rr.last_vmap = last_vmap;
    rr.dIdx = intensity_dx;
    rr.dIdy = intensity_dy;
    rr.pose = pose;
    rr.fx = K.fx;
    rr.fy = K.fy;
    rr.cx = K.cx;
    rr.cy = K.cy;
    rr.invfx = K.invfx;
    rr.invfy = K.invfy;
    rr.out = sum;

    rgb_reduce_kernel<<<96, 224>>>(rr);
    safe_call(hipDeviceSynchronize());
    safe_call(hipGetLastError());

    cv::cuda::reduce(sum, out, 0, cv::REDUCE_SUM);

    safe_call(hipDeviceSynchronize());
    safe_call(hipGetLastError());

    cv::Mat host_data;
    out.download(host_data);
    create_jtjjtr<6, 7>(host_data, jtj, jtr);
    residual[0] = host_data.ptr<float>()[27];
    residual[1] = host_data.ptr<float>()[28];
}

struct ICPReduction
{
    __device__ __inline__ bool searchPoint(int &x, int &y, float3 &vcurr_g, float3 &vlast_g, float3 &nlast_g) const
    {
        float3 vlast_c = make_float3(last_vmap_.ptr(y)[x]);
        if (isnan(vlast_c.x))
            return false;

        vlast_g = pose(vlast_c);

        float invz = 1.0 / vlast_g.z;
        int u = __float2int_rd(vlast_g.x * invz * fx + cx + 0.5);
        int v = __float2int_rd(vlast_g.y * invz * fy + cy + 0.5);
        if (u < 0 || v < 0 || u >= cols || v >= rows)
            return false;

        vcurr_g = make_float3(curr_vmap_.ptr(v)[u]);

        float3 nlast_c = make_float3(last_nmap_.ptr(y)[x]);
        nlast_g = pose.rotate(nlast_c);

        float3 ncurr_g = make_float3(curr_nmap_.ptr(v)[u]);

        float dist = norm(vlast_g - vcurr_g);
        float sine = norm(cross(ncurr_g, nlast_g));

        return (sine < angleThresh && dist <= distThresh && !isnan(ncurr_g.x) && !isnan(nlast_g.x));
    }

    __device__ __inline__ void getRow(int &i, float *sum) const
    {
        int y = i / cols;
        int x = i - y * cols;

        bool found = false;
        float3 vcurr, vlast, nlast;
        found = searchPoint(x, y, vcurr, vlast, nlast);
        float row[7] = {0, 0, 0, 0, 0, 0, 0};

        if (found)
        {
            *(float3 *)&row[0] = nlast;
            *(float3 *)&row[3] = cross(vlast, nlast);
            row[6] = nlast * (vcurr - vlast);
        }

        int count = 0;
#pragma unroll
        for (int i = 0; i < 7; ++i)
        {
#pragma unroll
            for (int j = i; j < 7; ++j)
                sum[count++] = row[i] * row[j];
        }

        sum[count] = (float)found;
    }

    __device__ __inline__ void operator()() const
    {
        float sum[29] = {0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0,
                         0, 0, 0, 0};

        int i = blockIdx.x * blockDim.x + threadIdx.x;
        float val[29];
        for (; i < N; i += blockDim.x * gridDim.x)
        {
            getRow(i, val);
#pragma unroll
            for (int j = 0; j < 29; ++j)
                sum[j] += val[j];
        }

        BlockReduce<float, 29>(sum);

        if (threadIdx.x == 0)
        {
#pragma unroll
            for (int i = 0; i < 29; ++i)
                out.ptr(blockIdx.x)[i] = sum[i];
        }
    }

    DeviceMatrix3x4 pose;
    cv::cuda::PtrStep<float4> curr_vmap_, last_vmap_;
    cv::cuda::PtrStep<float4> curr_nmap_, last_nmap_;
    int cols, rows, N;
    float fx, fy, cx, cy;
    float angleThresh, distThresh;
    mutable cv::cuda::PtrStepSz<float> out;
};

__global__ void icp_reduce_kernel(const ICPReduction icp)
{
    icp();
}

void icp_reduce(const cv::cuda::GpuMat &curr_vmap,
                const cv::cuda::GpuMat &curr_nmap,
                const cv::cuda::GpuMat &last_vmap,
                const cv::cuda::GpuMat &last_nmap,
                cv::cuda::GpuMat &sum,
                cv::cuda::GpuMat &out,
                const Sophus::SE3d &pose,
                const IntrinsicMatrix K,
                float *jtj, float *jtr,
                float *residual)
{
    int cols = curr_vmap.cols;
    int rows = curr_vmap.rows;

    ICPReduction icp;
    icp.out = sum;
    icp.curr_vmap_ = curr_vmap;
    icp.curr_nmap_ = curr_nmap;
    icp.last_vmap_ = last_vmap;
    icp.last_nmap_ = last_nmap;
    icp.cols = cols;
    icp.rows = rows;
    icp.N = cols * rows;
    icp.pose = pose;
    icp.angleThresh = cos(30 * 3.14 / 180);
    icp.distThresh = 0.01;
    icp.fx = K.fx;
    icp.fy = K.fy;
    icp.cx = K.cx;
    icp.cy = K.cy;

    icp_reduce_kernel<<<96, 224>>>(icp);
    cv::cuda::reduce(sum, out, 0, cv::REDUCE_SUM);

    cv::Mat host_data;
    out.download(host_data);
    create_jtjjtr<6, 7>(host_data, jtj, jtr);
    residual[0] = host_data.ptr<float>()[27];
    residual[1] = host_data.ptr<float>()[28];
}

} // namespace fusion